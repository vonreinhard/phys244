#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "../timer.h"

int main ( int argc, char *argv[] );
void compute ( int np, int nd, double pos[], double vel[], 
  double mass, double f[], double *pot, double *kin ,int j);
double dist ( int nd, double r1[], double r2[], double dr[] );
void initialize ( int np, int nd, double box[], int *seed, double pos[], 
  double vel[], double acc[] );
double r8_uniform_01 ( int *seed );
void timestamp ( );

#define gridSize 4
#define blockSize 1024
/*********************************/
// Do the update 
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         force, the force
//         mass, the mass of particle
//         dt,  the difference of time
// Output:
//         pos,  particle posiion
//         vel,  particle velocity
//         acc,  particle accelation rate
/*********************************/
__global__ void update ( int np, int nd, double* pos, double* vel, double* f, double* acc, double mass, double dt )
{
 
  double rmass;

  rmass = 1.0 / mass;



  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  while(idx < (np*nd) ){

    pos[idx] = pos[idx] + vel[idx] * dt + 0.5 * acc[idx] * dt * dt;
    vel[idx] = vel[idx] + 0.5 * dt * ( f[idx] * rmass + acc[idx] );
    acc[idx] = f[idx] * rmass;
    idx+=stride;
  }

  return;
}
/*********************************/
// Compute rij and d for later uses
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         force, the force
//         pos,  particle posiion
// Output:
//         d  particle d
//         rij,  distance from this position to j postion
/*********************************/
__global__ void compute_rd ( int np, int nd, double* pos,int j,double *d,double *rij){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  int k;
  if(idx>=np*nd)return;

  while(idx<np*nd){
    k = idx / nd;
    if(k!=j){
      rij[idx] = pos[idx] - pos[idx-nd*(k-j)];
      d[idx] = rij[idx]*rij[idx];

    }else{
      d[idx] = 0;
    }
    idx+=stride;
  }
}
/*********************************/
// Compute d2 and pe
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         d,  particle d
// Output:
//         d2,  based on d
//         pe,  potential for this situation
/*********************************/
__global__ void compute_d2 ( int np, int nd,double *d,double *d2,double *pe){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double PI2 = 3.141592653589793 / 2.0;
  int stride = gridDim.x*blockDim.x;
  if(idx>=np)return;
  while(idx<np){
    
    d[idx*nd] += d[idx*nd+1]+d[idx*nd+2];
    d[idx*nd] = sqrt( d[idx*nd]);
    if ( d[idx*nd] < PI2 ){
      d2[idx] = d[idx*nd];
    }else{
      d2[idx] = PI2;
    }
    pe[idx] =  0.5 * pow ( sin ( d2[idx] ), 2 );
    
    idx+=stride;
  }
}
/*********************************/
// Update force
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         d,  particle d
//         d2,  based on d
//         rij,  distance from this position to j postion
//         j, current j
// Output:
//         
//         force, the force of particle
/*********************************/
__global__ void compute_f ( int np, int nd,double *d,double *d2,double *f,double *rij,int j){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  int k;
  if(idx>=np*nd)return;
  while(idx<np*nd){

    k = idx / nd;
    if(k!=j){
      f[idx] -=  rij[idx] * sin ( 2.0 * d2[k] ) / d[k*nd];
    }
    // printf("%8f\n",sin ( 2.0 * d2[k] ));
    idx+=stride;
  }
  __syncthreads();
    
}
/*********************************/
// Reduction for pe
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         pe,  potential
//         j, current j
// Output:
//         
//         OUT, reduction result
/*********************************/
__global__  void add_pe(double *pe,int np,int nd,double *OUT,int j){
  __shared__ double sdata[1000];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x+tid;
 sdata[tid] = 0;
 if(i>=np*nd)return;
 if(gridDim.x!=1){
   while(i<np*nd){
      sdata[tid] += pe[i];
      i += gridDim.x*blockDim.x;
   }
  }else{
   sdata[tid] = pe[tid];
 }
 __syncthreads();
 for(int s=1;s<blockDim.x;s*=2){
   if(tid%(2*s)==0){
     sdata[tid]+=sdata[tid+s];
   }
   __syncthreads();
 }
 if(tid==0)pe[blockIdx.x]=sdata[0];
 i = blockIdx.x*blockDim.x+tid;
 if(i==0){
  OUT[j] = pe[0];
 }
}
/*********************************/
// Reduction for ke
// Input :
//         np,  number of particle
//         nd,  number of dimension
//         ke,  potential
//         mass, particle mass
//         vel, particle velocity
//         j, current j
// Output:
//         ke[0],  reduction result
/*********************************/
__global__  void add_ke(double *ke,double* vel,int np,int nd, double mass){
  __shared__ double sdata[1000];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x+tid;
  sdata[tid] = 0;
 if(i>=np*nd)return;
 if(gridDim.x!=1){
   while(i<np*nd){
      sdata[tid] += vel[i]*vel[i];
      i += gridDim.x*blockDim.x;
   }
  }else{
   sdata[tid] = ke[tid];
 }
 __syncthreads();
 for(int s=1;s<blockDim.x;s*=2){
   if(tid%(2*s)==0){
     sdata[tid]+=sdata[tid+s];
   }
   __syncthreads();
 }
 if(tid==0)ke[blockIdx.x]=sdata[0];
 if(blockIdx.x*blockDim.x+tid==0&&gridDim.x==1)ke[0]*=0.5*mass;
}
int main ( int argc, char *argv[] )
{
  double *acc;
  double *box;
  double dt = 0.0001;
  double e0;
  double *force;
  int i;
  double kinetic;
  double mass = 1.0;
  int nd = 3;
  int np = 1000;
  double *pos;
  double potential;
  int seed = 123456789;
  int step;
  int step_num = 100;
  double *vel;

  timestamp ( );

  acc = ( double * ) malloc ( nd * np * sizeof ( double ) );
  box = ( double * ) malloc ( nd * sizeof ( double ) );
  force = ( double * ) malloc ( nd * np * sizeof ( double ) );
  pos = ( double * ) malloc ( nd * np * sizeof ( double ) );
  vel = ( double * ) malloc ( nd * np * sizeof ( double ) );

  
/*
  Set the dimensions of the box.
*/
  for ( i = 0; i < nd; i++ )
  {
    box[i] = 10.0;
  }

/*
  Set initial positions, velocities, and accelerations.
*/
  initialize ( np, nd, box, &seed, pos, vel, acc );
/*
  Compute the forces and energies.
*/
/*
  Allocate all value it will use at here
*/
  double* d_acc, *d_force,*d_pos,*d_vel,*ke,*d,*d2,*pe,*rij,*sumpe;
  hipMalloc(&d_acc, nd * np * sizeof ( double ));
  hipMalloc(&d_force, nd * np * sizeof ( double ));
  hipMalloc(&d_pos, nd * np * sizeof ( double ));
  hipMalloc(&d_vel, nd * np * sizeof ( double ));
  hipMalloc(&ke, np *sizeof ( double ));
  hipMalloc(&rij, nd * np *sizeof ( double ));
  hipMalloc(&pe, np *sizeof ( double ));
  hipMalloc(&d, nd * np *sizeof ( double ));
  hipMalloc(&d2, np *sizeof ( double ));
  hipMalloc(&sumpe, np *sizeof ( double ));
/*
  For initialization we need to copy pos,vel,acc to device and set force as 0
*/
  hipMemset(d_force,0.0,nd * np * sizeof ( double ));
  hipMemcpy(d_pos, pos, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  hipMemcpy(d_vel, vel, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  hipMemcpy(d_acc, acc, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  potential = 0.0;
/*
  Parallelize it under j loop
*/
  for(int j=0;j<np;j++){
/*
  Do compute
*/
    compute_rd<< <gridSize, blockSize >> > (np, nd, d_pos, j, d,rij);
    compute_d2 << <gridSize, blockSize >> >(np, nd, d, d2, pe);
    compute_f<< <gridSize, blockSize >> >  (np,nd,d,d2,d_force,rij,j);
/*
  pe add reduction
*/
    add_pe<< <gridSize, blockSize >> >(pe,1,np,sumpe,j);
  }
/*
  pe add reduction
*/
  add_pe<< <gridSize, blockSize >> >(sumpe,1,np,sumpe,0);
  double tmp_pe;
  hipMemcpy(&tmp_pe, sumpe, sizeof ( double ), hipMemcpyDeviceToHost);
  potential = tmp_pe;
/*
  ke add reduction
*/
  add_ke<< <gridSize, blockSize >> >(ke,d_vel,np,nd,mass);
  if(gridSize>1)
    add_ke<< <1, blockSize >> >(ke,ke,1,blockSize,mass);
  double tmp;
  hipMemcpy(&tmp, ke, sizeof ( double ), hipMemcpyDeviceToHost);
  kinetic = tmp;
  
  e0 = potential + kinetic;
 
  StartTimer();;
  
  
  for ( step = 1; step <= step_num; step++ )
  {
    
    hipMemset(d_force,0.0000000,nd * np * sizeof ( double ));
    
    potential = 0.0;
    for(int j=0;j<np;j++){


      compute_rd<< <gridSize, blockSize >> > (np, nd, d_pos, j, d,rij);
      compute_d2 << <gridSize, blockSize >> >(np, nd, d, d2, pe);
      compute_f<< <gridSize, blockSize >> >  (np,nd,d,d2,d_force,rij,j);

      add_pe<< <gridSize, blockSize >> >(pe,1,np,sumpe,j);
    }
 /*
  pe add reduction
*/   
    add_pe<< <gridSize, blockSize >> >(sumpe,1,np,sumpe,0);
    double tmp_pe;
    hipMemcpy(&tmp_pe, sumpe, sizeof ( double ), hipMemcpyDeviceToHost);
    potential = tmp_pe;
 /*
  ke add reduction
*/   
  
    add_ke<< <gridSize, blockSize >> >(ke,d_vel,np,nd,mass);
    if(gridSize>1)
      add_ke<< <1, blockSize >> >(ke,ke,1,blockSize,mass);
    double tmp;
    hipMemcpy(&tmp, ke, sizeof ( double ), hipMemcpyDeviceToHost);
    kinetic = tmp;
    
   
/*
  do updation
*/ 
    update<< <gridSize, blockSize >> > ( np, nd, d_pos, d_vel, d_force, d_acc, mass, dt );
 
  }
  double runtime = GetTimer();

  printf ( "\n" );
  printf ( "  Elapsed time for main computation:\n" );
  printf ( "  %f seconds.\n", runtime/1000 );
/*
  Free memory.
*/
  free ( acc );
  free ( box );
  free ( force );
  free ( pos );
  free ( vel );
/*
  Free cuda memory.
*/ 
  hipFree ( d_acc );
  hipFree ( d_force );
  hipFree ( d_pos );
  hipFree ( d_vel );
  hipFree ( ke );
  hipFree ( rij );
  hipFree ( pe );
  hipFree ( d );
  hipFree ( d2 );


  
/*
  Terminate.
*/
  printf ( "\n" );
  printf ( "  Normal end of execution.\n" );
  printf ( "\n" );
  timestamp ( );

  return 0;
}
/******************************************************************************/
/******************************************************************************/

void initialize ( int np, int nd, double box[], int *seed, double pos[], 
  double vel[], double acc[] )

/******************************************************************************/
/*
  Purpose:

    INITIALIZE initializes the positions, velocities, and accelerations.

  Licensing:

    This code is distributed under the GNU LGPL license. 

  Modified:

    21 November 2007

  Author:

    Original FORTRAN77 version by Bill Magro.
    C version by John Burkardt.

  Parameters:

    Input, int NP, the number of particles.

    Input, int ND, the number of spatial dimensions.

    Input, double BOX[ND], specifies the maximum position
    of particles in each dimension.

    Input, int *SEED, a seed for the random number generator.

    Output, double POS[ND*NP], the position of each particle.

    Output, double VEL[ND*NP], the velocity of each particle.

    Output, double ACC[ND*NP], the acceleration of each particle.
*/
{
  int i;
  int j;
/*
  Give the particles random positions within the box.
*/
  for ( i = 0; i < nd; i++ )
  {
    for ( j = 0; j < np; j++ )
    {
      pos[i+j*nd] = box[i] * r8_uniform_01 ( seed );
    }
  }

  for ( j = 0; j < np; j++ )
  {
    for ( i = 0; i < nd; i++ )
    {
      vel[i+j*nd] = 0.0;
    }
  }
  for ( j = 0; j < np; j++ )
  {
    for ( i = 0; i < nd; i++ )
    {
      acc[i+j*nd] = 0.0;
    }
  }
  return;
}
/******************************************************************************/

double r8_uniform_01 ( int *seed )

/******************************************************************************/
/*
  Purpose:

    R8_UNIFORM_01 is a unit pseudorandom R8.

  Discussion:

    This routine implements the recursion

      seed = 16807 * seed mod ( 2**31 - 1 )
      unif = seed / ( 2**31 - 1 )

    The integer arithmetic never requires more than 32 bits,
    including a sign bit.

  Licensing:

    This code is distributed under the GNU LGPL license. 

  Modified:

    11 August 2004

  Author:

    John Burkardt

  Reference:

    Paul Bratley, Bennett Fox, Linus Schrage,
    A Guide to Simulation,
    Springer Verlag, pages 201-202, 1983.

    Bennett Fox,
    Algorithm 647:
    Implementation and Relative Efficiency of Quasirandom
    Sequence Generators,
    ACM Transactions on Mathematical Software,
    Volume 12, Number 4, pages 362-376, 1986.

  Parameters:

    Input/output, int *SEED, a seed for the random number generator.

    Output, double R8_UNIFORM_01, a new pseudorandom variate, strictly between
    0 and 1.
*/
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * ( *seed - k * 127773 ) - k * 2836;

  if ( *seed < 0 )
  {
    *seed = *seed + 2147483647;
  }

  r = ( double ) ( *seed ) * 4.656612875E-10;

  return r;
}
/******************************************************************************/

void timestamp ( void )
{
# define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  time_t now;

  now = time ( NULL );
  tm = localtime ( &now );

  strftime ( time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm );

  printf ( "%s\n", time_buffer );

  return;
# undef TIME_SIZE
}
/******************************************************************************/
