#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "../timer.h"

int main ( int argc, char *argv[] );
void compute ( int np, int nd, double pos[], double vel[], 
  double mass, double f[], double *pot, double *kin ,int j);
double dist ( int nd, double r1[], double r2[], double dr[] );
void initialize ( int np, int nd, double box[], int *seed, double pos[], 
  double vel[], double acc[] );
double r8_uniform_01 ( int *seed );
void timestamp ( );

#define gridSize 4
#define blockSize 1024
__global__ void update ( int np, int nd, double* pos, double* vel, double* f, double* acc, double mass, double dt )
{
 
  double rmass;

  rmass = 1.0 / mass;



  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  // printf("%8d\n",idx);
  while(idx < (np*nd) ){

    pos[idx] = pos[idx] + vel[idx] * dt + 0.5 * acc[idx] * dt * dt;
    vel[idx] = vel[idx] + 0.5 * dt * ( f[idx] * rmass + acc[idx] );
    acc[idx] = f[idx] * rmass;
    idx+=stride;
  }

  return;
}
/*********************************/
__global__ void compute_rd ( int np, int nd, double* pos,int j,double *d,double *rij){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  int k;
  if(idx>=np*nd)return;

  while(idx<np*nd){
    k = idx / nd;
    if(k!=j){
      rij[idx] = pos[idx] - pos[idx-nd*(k-j)];
      d[idx] = rij[idx]*rij[idx];
      


    }else{
      d[idx] = 0;
    }
    idx+=stride;
  }
}
__global__ void compute_d2 ( int np, int nd,double *d,double *d2,double *pe){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  double PI2 = 3.141592653589793 / 2.0;
  int stride = gridDim.x*blockDim.x;
  if(idx>=np)return;
  while(idx<np){
    
    d[idx*nd] += d[idx*nd+1]+d[idx*nd+2];
    d[idx*nd] = sqrt( d[idx*nd]);
    if ( d[idx*nd] < PI2 ){
      d2[idx] = d[idx*nd];
    }else{
      d2[idx] = PI2;
    }
    pe[idx] =  0.5 * pow ( sin ( d2[idx] ), 2 );
    
    idx+=stride;
  }
}
__global__ void compute_f ( int np, int nd,double *d,double *d2,double *f,double *rij,int j){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x*blockDim.x;
  int k;
  if(idx>=np*nd)return;
  while(idx<np*nd){

    k = idx / nd;
    if(k!=j){
      f[idx] -=  rij[idx] * sin ( 2.0 * d2[k] ) / d[k*nd];
    }
    // printf("%8f\n",sin ( 2.0 * d2[k] ));
    idx+=stride;
  }
  __syncthreads();
    
}
/********************************************************************************/
__global__  void add_pe(double *pe,int np,int nd,double *OUT,int j){
  __shared__ double sdata[1000];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x+tid;
 sdata[tid] = 0;
 if(i>=np*nd)return;
 if(gridDim.x!=1){
   while(i<np*nd){
      sdata[tid] += pe[i];
      i += gridDim.x*blockDim.x;
   }
  }else{
   sdata[tid] = pe[tid];
 }
 __syncthreads();
 for(int s=blockDim.x/2;s>0;s/=2){
  if(tid<s){
    sdata[tid]+=sdata[tid+s];
  }
   __syncthreads();
 }
 if(tid==0)pe[blockIdx.x]=sdata[0];
 i = blockIdx.x*blockDim.x+tid;
 if(i==0){
  OUT[j] = pe[0];
 }
}
// compute ke;
/********************************************************************************/
__global__  void add_ke(double *ke,double* vel,int np,int nd, double mass){
  __shared__ double sdata[1000];
 int tid = threadIdx.x;
 int i = blockIdx.x*blockDim.x+tid;
//  printf("%8d\n",i);
  sdata[tid] = 0;
 if(i>=np*nd)return;
 if(gridDim.x!=1){
   while(i<np*nd){
      sdata[tid] += vel[i]*vel[i];
      i += gridDim.x*blockDim.x;
   }
  }else{
   sdata[tid] = ke[tid];
 }
 __syncthreads();
 for(int s=blockDim.x/2;s>0;s/=2){
  if(tid<s){
    sdata[tid]+=sdata[tid+s];
  }
   __syncthreads();
 }
 if(tid==0)ke[blockIdx.x]=sdata[0];
 if(blockIdx.x*blockDim.x+tid==0&&gridDim.x==1)ke[0]*=0.5*mass;
}

/******************************************************************************/
void outputval(double *val,int np,int nd){
  for(int i=0;i<np;i++){
    for(int j=0;j<nd;j++){
      printf("%4f ",val[i*nd+j]);
    }
    printf("\n");
  }
}
int main ( int argc, char *argv[] )
{
  double *acc;
  double *box;
  double dt = 0.0001;
  double e0;
  double *force;
  int i;
  double kinetic;
  double mass = 1.0;
  int nd = 3;
  int np = 1000;
  double *pos;
  double potential;
  int seed = 123456789;
  int step;
  int step_num = 100;
  int step_print;
  int step_print_index;
  int step_print_num;
  double *vel;
  // double wtime;

  timestamp ( );

  acc = ( double * ) malloc ( nd * np * sizeof ( double ) );
  box = ( double * ) malloc ( nd * sizeof ( double ) );
  force = ( double * ) malloc ( nd * np * sizeof ( double ) );
  pos = ( double * ) malloc ( nd * np * sizeof ( double ) );
  vel = ( double * ) malloc ( nd * np * sizeof ( double ) );

  printf ( "\n" );
  printf ( "MD_OPENMP\n" );
  printf ( "  C/OpenMP version\n" );
  printf ( "  A molecular dynamics program.\n" );

  printf ( "\n" );
  printf ( "  NP, the number of particles in the simulation is %d\n", np );
  printf ( "  STEP_NUM, the number of time steps, is %d\n", step_num );
  printf ( "  DT, the size of each time step, is %f\n", dt );

  
/*
  Set the dimensions of the box.
*/
  for ( i = 0; i < nd; i++ )
  {
    box[i] = 10.0;
  }

  printf ( "\n" );
  printf ( "  Initializing positions, velocities, and accelerations.\n" );
/*
  Set initial positions, velocities, and accelerations.
*/
  initialize ( np, nd, box, &seed, pos, vel, acc );
/*
  Compute the forces and energies.
*/
  printf ( "\n" );
  printf ( "  Computing initial forces and energies.\n" );
  // memalloc
  double* d_acc, *d_force,*d_pos,*d_vel,*ke,*d,*d2,*pe,*rij,*sumpe;
  hipMalloc(&d_acc, nd * np * sizeof ( double ));
  hipMalloc(&d_force, nd * np * sizeof ( double ));
  hipMalloc(&d_pos, nd * np * sizeof ( double ));
  hipMalloc(&d_vel, nd * np * sizeof ( double ));
  hipMalloc(&ke, np *sizeof ( double ));
  hipMalloc(&rij, nd * np *sizeof ( double ));
  hipMalloc(&pe, np *sizeof ( double ));
  hipMalloc(&d, nd * np *sizeof ( double ));
  hipMalloc(&d2, np *sizeof ( double ));
  hipMalloc(&sumpe, np *sizeof ( double ));
  // compute sth
  hipMemset(d_force,0.0,nd * np * sizeof ( double ));
  hipMemcpy(d_pos, pos, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  hipMemcpy(d_vel, vel, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  hipMemcpy(d_acc, acc, nd * np * sizeof ( double ), hipMemcpyHostToDevice);
  // outputval(force,np,nd);
  potential = 0.0;
  // double total_pe = 0.0;
  for(int j=0;j<np;j++){

    compute_rd<< <gridSize, blockSize >> > (np, nd, d_pos, j, d,rij);
    compute_d2 << <gridSize, blockSize >> >(np, nd, d, d2, pe);
    compute_f<< <gridSize, blockSize >> >  (np,nd,d,d2,d_force,rij,j);

    add_pe<< <gridSize, blockSize >> >(pe,1,np,sumpe,j);
    
    // double tmp_pe;
    // hipMemcpy(&tmp_pe, pe, sizeof ( double ), hipMemcpyDeviceToHost);
    // total_pe += tmp_pe;
  }
  add_pe<< <gridSize, blockSize >> >(sumpe,1,np,sumpe,0);
  double tmp_pe;
  hipMemcpy(&tmp_pe, sumpe, sizeof ( double ), hipMemcpyDeviceToHost);
  printf("%8f\n",tmp_pe);
  potential = tmp_pe;

  add_ke<< <gridSize, blockSize >> >(ke,d_vel,np,nd,mass);
  if(gridSize>1)
    add_ke<< <1, blockSize >> >(ke,ke,1,blockSize,mass);
  double tmp;
  hipMemcpy(&tmp, ke, sizeof ( double ), hipMemcpyDeviceToHost);
  kinetic = tmp;
  
  e0 = potential + kinetic;
  
/*
  This is the main time stepping loop:
    Compute forces and energies,
    Update positions, velocities, accelerations.
*/
  printf ( "\n" );
  printf ( "  At each step, we report the potential and kinetic energies.\n" );
  printf ( "  The sum of these energies should be a constant.\n" );
  printf ( "  As an accuracy check, we also print the relative error\n" );
  printf ( "  in the total energy.\n" );
  printf ( "\n" );
  printf ( "      Step      Potential       Kinetic        (P+K-E0)/E0\n" );
  printf ( "                Energy P        Energy K       Relative Energy Error\n" );
  printf ( "\n" );

  step_print = 0;
  step_print_index = 0;
  step_print_num = 10;
  
  step = 0;
  printf ( "  %8d  %14f  %14f  %14e\n",
    step, potential, kinetic, ( potential + kinetic - e0 ) / e0 );
  step_print_index = step_print_index + 1;
  step_print = ( step_print_index * step_num ) / step_print_num;

  StartTimer();;
  // parameter initialization
  
  
  for ( step = 1; step <= step_num; step++ )
  {
    
    hipMemset(d_force,0.0000000,nd * np * sizeof ( double ));
    // hipMemcpy(force, d_force, nd * np * sizeof ( double ), hipMemcpyDeviceToHost);
    
    potential = 0.0;
    for(int j=0;j<np;j++){


      compute_rd<< <gridSize, blockSize >> > (np, nd, d_pos, j, d,rij);
      compute_d2 << <gridSize, blockSize >> >(np, nd, d, d2, pe);
      compute_f<< <gridSize, blockSize >> >  (np,nd,d,d2,d_force,rij,j);

      add_pe<< <gridSize, blockSize >> >(pe,1,np,sumpe,j);
      // tmp_pe<< <    1   ,      1    >> >(sumpe,pe,j);
      // double tmp_pe;
      // hipMemcpy(&tmp_pe, pe, sizeof ( double ), hipMemcpyDeviceToHost);
      // total_pe += tmp_pe;

    }
    add_pe<< <gridSize, blockSize >> >(sumpe,1,np,sumpe,0);
    double tmp_pe;
    hipMemcpy(&tmp_pe, sumpe, sizeof ( double ), hipMemcpyDeviceToHost);
    // double *f = ( double * ) malloc ( nd * np * sizeof ( double ) );
    //   hipMemcpy(f, d_force, np*nd*sizeof ( double ), hipMemcpyDeviceToHost);
    // outputval(f,np,nd);
    potential = tmp_pe;
    
  
    
    
    

    // compute ke
    add_ke<< <gridSize, blockSize >> >(ke,d_vel,np,nd,mass);
    if(gridSize>1)
      add_ke<< <1, blockSize >> >(ke,ke,1,blockSize,mass);
    double tmp;
    hipMemcpy(&tmp, ke, sizeof ( double ), hipMemcpyDeviceToHost);
    kinetic = tmp;
    if ( step == step_print )
    {
      printf ( "  %8d  %14f  %14f  %14e\n",
    step, potential, kinetic, ( potential + kinetic - e0 ) / e0 );
      step_print_index = step_print_index + 1;
      step_print = ( step_print_index * step_num ) / step_print_num;
    }
   

    update<< <gridSize, blockSize >> > ( np, nd, d_pos, d_vel, d_force, d_acc, mass, dt );
 
  }
  double runtime = GetTimer();

  printf ( "\n" );
  printf ( "  Elapsed time for main computation:\n" );
  printf ( "  %f seconds.\n", runtime/1000 );
/*
  Free memory.
*/
  free ( acc );
  free ( box );
  free ( force );
  free ( pos );
  free ( vel );
  // cuda
  hipFree ( d_acc );
  hipFree ( d_force );
  hipFree ( d_pos );
  hipFree ( d_vel );
  hipFree ( ke );
  hipFree ( rij );
  hipFree ( pe );
  hipFree ( d );
  hipFree ( d2 );


  
/*
  Terminate.
*/
  printf ( "\n" );
  printf ( "MD_OPENMP\n" );
  printf ( "  Normal end of execution.\n" );
  printf ( "\n" );
  timestamp ( );

  return 0;
}
/******************************************************************************/
/******************************************************************************/

void initialize ( int np, int nd, double box[], int *seed, double pos[], 
  double vel[], double acc[] )

/******************************************************************************/
/*
  Purpose:

    INITIALIZE initializes the positions, velocities, and accelerations.

  Licensing:

    This code is distributed under the GNU LGPL license. 

  Modified:

    21 November 2007

  Author:

    Original FORTRAN77 version by Bill Magro.
    C version by John Burkardt.

  Parameters:

    Input, int NP, the number of particles.

    Input, int ND, the number of spatial dimensions.

    Input, double BOX[ND], specifies the maximum position
    of particles in each dimension.

    Input, int *SEED, a seed for the random number generator.

    Output, double POS[ND*NP], the position of each particle.

    Output, double VEL[ND*NP], the velocity of each particle.

    Output, double ACC[ND*NP], the acceleration of each particle.
*/
{
  int i;
  int j;
/*
  Give the particles random positions within the box.
*/
  for ( i = 0; i < nd; i++ )
  {
    for ( j = 0; j < np; j++ )
    {
      pos[i+j*nd] = box[i] * r8_uniform_01 ( seed );
    }
  }

  for ( j = 0; j < np; j++ )
  {
    for ( i = 0; i < nd; i++ )
    {
      vel[i+j*nd] = 0.0;
    }
  }
  for ( j = 0; j < np; j++ )
  {
    for ( i = 0; i < nd; i++ )
    {
      acc[i+j*nd] = 0.0;
    }
  }
  return;
}
/******************************************************************************/

double r8_uniform_01 ( int *seed )

/******************************************************************************/
/*
  Purpose:

    R8_UNIFORM_01 is a unit pseudorandom R8.

  Discussion:

    This routine implements the recursion

      seed = 16807 * seed mod ( 2**31 - 1 )
      unif = seed / ( 2**31 - 1 )

    The integer arithmetic never requires more than 32 bits,
    including a sign bit.

  Licensing:

    This code is distributed under the GNU LGPL license. 

  Modified:

    11 August 2004

  Author:

    John Burkardt

  Reference:

    Paul Bratley, Bennett Fox, Linus Schrage,
    A Guide to Simulation,
    Springer Verlag, pages 201-202, 1983.

    Bennett Fox,
    Algorithm 647:
    Implementation and Relative Efficiency of Quasirandom
    Sequence Generators,
    ACM Transactions on Mathematical Software,
    Volume 12, Number 4, pages 362-376, 1986.

  Parameters:

    Input/output, int *SEED, a seed for the random number generator.

    Output, double R8_UNIFORM_01, a new pseudorandom variate, strictly between
    0 and 1.
*/
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * ( *seed - k * 127773 ) - k * 2836;

  if ( *seed < 0 )
  {
    *seed = *seed + 2147483647;
  }

  r = ( double ) ( *seed ) * 4.656612875E-10;

  return r;
}
/******************************************************************************/

void timestamp ( void )
{
# define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  time_t now;

  now = time ( NULL );
  tm = localtime ( &now );

  strftime ( time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm );

  printf ( "%s\n", time_buffer );

  return;
# undef TIME_SIZE
}
/******************************************************************************/
